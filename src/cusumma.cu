#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

extern "C" void
cusumma(unsigned int transA, 
        unsigned int transB, 
        unsigned int m,
        unsigned int n,
        unsigned int k,
        float *A, 
        float *B, 
        float *C)
{
  float *hA, *dA, *dB, *dC;
  int i, j, diff, tm, tk, tp, tp_last, tmp1, tmp2;
  char opA, opB;
  float factor;
  unsigned int gpu_mem, _m, _mmax, _moff, _k, _kmax, _koff;

  cublasInit();

  // get total free memory available to SUMMA
  hipMemGetInfo(&gpu_mem, NULL);

  // take 2MB off the top for CUBLAS working memory
  // this is a guess that seems to work, replace with actual numbers when known
  gpu_mem -= 2*1048576;

  // convert gpu_mem from bytes into matrix elements (floats) for simplicity
  gpu_mem /= sizeof(float);

  // determine optimal partition dimensions
  tp = 100000;
  tm = 0;
  do {
    if(tp > 0)
      tp_last = tp;
    _mmax = ceil(1.0*m/++tm);
    tmp1 = A == B ? gpu_mem - _mmax * _mmax : gpu_mem - n * _mmax;
    tmp2 = A == B ? _mmax : n + _mmax;
    _kmax = tmp1 / tmp2; //(gpu_mem - n * _mmax)/(n + _mmax);
    tk    = ceil(1.0*k/_kmax);
    tp    = (A == A ? 1 : 2)*tm*tk + tm;
  } while(tp < 0 || tp < tp_last);

  _mmax = ceil(1.0*m/--tm);
  if(A == B) {
    _kmax = gpu_mem / _mmax - _mmax;
  } else {
    _kmax = ( gpu_mem - _mmax * n ) / ( _mmax + n );
  }
 
//_mmax = 2;
//_kmax = 2;

  // assumes input matrices are in row-major order
  opA = transB ? 't' : 'n';
  opB = transA ? 't' : 'n';

  _m    = _mmax;
  _moff = 0;
  while(_moff < m) {
    cublasAlloc(_m * n, sizeof(float), (void**)&dC);
    if(A == B) {  // op(A) * op(A)

      diff = gpu_mem - m*k - m*m;
      // A can fit entirely on the device
      if((_m == m) && (diff > 0)) {
        cublasAlloc(m * k, sizeof(float), (void**)&dA);
        hipblasSetVector(m * k, sizeof(float), A, 1, dA, 1);
        
        hipblasSgemm(opA, opB, m, m, k, 1.0f, dA, k, dA, k, 0.0f, dC, m);
        cublasFree(dA);

      } else {
        _koff  = 0;
        _k     = _kmax;
        factor = 0.0f;
        while(_koff < k) {
          cublasAlloc(_m * _k, sizeof(float), (void**)&dA);

          hA = (float*)malloc(_m * _k * sizeof(float));
          for(i = 0; i < _m; ++i)
            for(j = 0; j < _k; ++j)
              hA[i*_k + j] = A[(i+_moff)*k + j + _koff];
          hipblasSetVector(_m * _k, sizeof(float), hA, 1, dA, 1);
          free(hA);

          hipblasSgemm(opA, opB, _m, _m, _k, 1.0f, dA, _k, dA, _k, factor, dC, _m);
          cublasFree(dA);
        
          _koff += _k;
          _k     = k - _koff > _kmax ? _kmax : k - _koff;
          factor = 1.0f;
        }
      }

    } else { // op(A) * op(B)
      cublasAlloc(_m * n, sizeof(float), (void**)&dC);
      diff = gpu_mem - (m*k + k*n + m*n);
      if((_m == m) && (diff > 0)) {
        cublasAlloc(m * k, sizeof(float), (void**)&dA);
        hipblasSetVector(m * k, sizeof(float), A, 1, dA, 1);

        cublasAlloc(k * n, sizeof(float), (void**)&dB);
        hipblasSetVector(k * n, sizeof(float), B, 1, dB, 1);
    
        hipblasSgemm(opA, opB, n, m, k, 1.0f, dB, n, dA, k, 0.0f, dC, m);

        cublasFree(dA);
        cublasFree(dB);

      } else {
        _koff  = 0;
        _k     = _kmax;
        factor = 0.0f;
        while(_koff < k) {
          cublasAlloc(_m * _k, sizeof(float), (void**)&dA);
          cublasAlloc(_k * n, sizeof(float), (void**)&dB);

          hA = (float*) malloc(_m * _k * sizeof(float));
          for(i = 0; i < _m; ++i)
            for(j = 0; j < _k; ++j)
              hA[i*_k + j] = A[(i+_moff)*k + j + _koff];
          hipblasSetVector(_m * _k, sizeof(float), hA, 1, dA, 1);
          free(hA);
/*
        //hB = (float*) malloc(_k * n * sizeof(float));
        for(i = 0; i < _k; ++i)
          for(j = 0; j < n; ++j)
            hB[i*n + j] = B[(i+_koff)*n + j];
        //hipblasSetMatrix(_k, n, sizeof(float), hB, _k, dB, _k);
        hipblasSetVector(_k * n, sizeof(float), hB, 1, dB, 1);
        free(hB);
*/
          int ldb = transB ? _k : n;
          hipblasSetVector(_k * n, sizeof(float), B+(n*_koff), 1, dB, 1);
          hipblasSgemm(opA, opB, n, _m, _k, 1.0f, dB, ldb, dA, _k, factor, dC, n);
          cublasFree(dA);
          cublasFree(dB);

          _koff += _k;
          _k = k - _koff > _kmax ? _kmax : k - _koff;
          factor = 1.0f;
        }
      }
    }

    hipblasGetVector(_m*n, sizeof(float), dC, 1, C+(_moff*n), 1);
    cublasFree(dC);

    _moff += _m;
    _m = m - _moff > _mmax ? _mmax : m - _moff;
  }

  cublasShutdown();
}

int main(int argc, char** argv) {
  struct timeval start, end; 
  double elapsed;

  float *A, *B, *C;
  int i, m, n, k;
 
  m = 500;
  n = 500;
  k = 400000;
  A = (float*)malloc(m*k*sizeof(float));
  B = (float*)malloc(k*n*sizeof(float));
  C = (float*)calloc(m*n,sizeof(float));

  for(i = 0; i < m*k; ++i)
    A[i] = 1;
  for(i = 0; i < k*n; ++i)
    B[i] = 1;

for(i=0;i<11;++i) {
  gettimeofday(&start,NULL);
  cusumma(0,1,m,n,k,A,B,C);
  gettimeofday(&end,NULL);

  elapsed = ((end.tv_sec*1000000 + end.tv_usec) - (start.tv_sec*1000000 + start.tv_usec))/1000000.0;
  printf("%f %f %f %f\n", C[0], C[m-1], C[m*(n-1)], C[m*n-1]);
//  for(i = 0; i < m*m; ++i)
//    printf("%f\n",C[i]);
  printf("%f\n", elapsed);
}

  free(A);
  free(B);
  free(C);
}
