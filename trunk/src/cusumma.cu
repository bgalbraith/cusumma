#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

extern "C" void
cusumma(unsigned int transA, 
        unsigned int transB, 
        unsigned int m,
        unsigned int n,
        unsigned int k,
        float *A, 
        float *B, 
        float *C)
{
  float *hA, *dA, *dB, *dC;
  int i, j, diff, tm, tk, tp, tp_last, tmp1, tmp2, _kmax, _mmax, _m, _moff, _k, _koff;
  char opA, opB;
  float factor;
  unsigned int gpu_mem, lda, ldb;

  cublasInit();

  // get total free memory available to CUSUMMA
  hipMemGetInfo(&gpu_mem, NULL);

  // take 2MB off the top for CUBLAS working memory
  // this is a guess that seems to work, replace with actual numbers when known
  gpu_mem -= 2*1048576;

  // convert gpu_mem from bytes into matrix elements (floats) for simplicity
  gpu_mem /= sizeof(float);

  // determine optimal partition dimensions
  tp = 100000;
  tm = 0;
  do {
    if(tp > 0)
      tp_last = tp;
    _mmax = ceil(1.0*m/++tm);
    tmp1 = gpu_mem - _mmax * (A == B ? _mmax : n);
    tmp2 = _mmax + (A == B ? 0 : n);
    _kmax = tmp1 / tmp2; //(gpu_mem - n * _mmax)/(n + _mmax);
    tk    = ceil(1.0*k / _kmax);
    tp    = (A == B ? 1 : 2)*tm*tk + tm;
  } while(tp < 0 || tp < tp_last);

  _mmax = ceil(1.0*m/--tm);
  if(A == B) {
    _kmax = gpu_mem / _mmax - _mmax;
  } else {
    _kmax = (gpu_mem - _mmax*n )/(_mmax + n);
  }
//_mmax = 2;
//_kmax = 2;

  // assumes input matrices are in row-major order
  opA = transB ? 't' : 'n';
  opB = transA ? 't' : 'n';

  _m    = _mmax;
  _moff = 0;
  while(_moff < m) {
    cublasAlloc(_m * n, sizeof(float), (void**)&dC);
    if(A == B) {  // op(A) * op(A)
      diff = gpu_mem - m*k - m*m;
      if((_m == m) && (diff > 0)) {
        cublasAlloc(m * k, sizeof(float), (void**)&dA);
        hipblasSetVector(m * k, sizeof(float), A, 1, dA, 1);

        lda = transA ? m : k;
        ldb = transB ? k : n;
        hipblasSgemm(opA, opB, n, m, k, 1.0f, dA, ldb, dA, lda, 0.0f, dC, n);
        cublasFree(dA);

      } else {
        _koff  = 0;
        _k     = _kmax;
        factor = 0.0f;
        while(_koff < k) {
          cublasAlloc(_m * _k, sizeof(float), (void**)&dA);

          hA = (float*)malloc(_m * _k * sizeof(float));
          for(i = 0; i < _m; ++i)
            for(j = 0; j < _k; ++j)
              hA[i*_k + j] = A[(i+_moff)*k + j + _koff];
          hipblasSetVector(_m * _k, sizeof(float), hA, 1, dA, 1);
          free(hA);

          lda = transA ? _m : _k;
          ldb = transB ? _k : n;
          hipblasSgemm(opA, opB, n, _m, _k, 1.0f, dA, ldb, dA, lda, factor, dC, n);
          cublasFree(dA);
        
          _koff += _k;
          _k     = k - _koff > _kmax ? _kmax : k - _koff;
          factor = 1.0f;
        }
      }

    } else { // op(A) * op(B)
      diff = gpu_mem - (m*k + k*n + m*n);
      if((_m == m) && (diff > 0)) {
        cublasAlloc(m * k, sizeof(float), (void**)&dA);
        hipblasSetVector(m * k, sizeof(float), A, 1, dA, 1);

        cublasAlloc(k * n, sizeof(float), (void**)&dB);
        hipblasSetVector(k * n, sizeof(float), B, 1, dB, 1);
    
        lda = transA ? m : k;
        ldb = transB ? k : n;
        hipblasSgemm(opA, opB, n, m, k, 1.0f, dB, ldb, dA, lda, 0.0f, dC, n);

        cublasFree(dA);
        cublasFree(dB);

      } else {
        _koff  = 0;
        _k     = _kmax;
        factor = 0.0f;
        while(_koff < k) {
          cublasAlloc(_m * _k, sizeof(float), (void**)&dA);
          cublasAlloc(_k * n, sizeof(float), (void**)&dB);

          hA = (float*) malloc(_m * _k * sizeof(float));
          for(i = 0; i < _m; ++i)
            for(j = 0; j < _k; ++j)
              hA[i*_k + j] = A[(i+_moff)*k + j + _koff];
          hipblasSetVector(_m * _k, sizeof(float), hA, 1, dA, 1);
          free(hA);
/*
        //hB = (float*) malloc(_k * n * sizeof(float));
        for(i = 0; i < _k; ++i)
          for(j = 0; j < n; ++j)
            hB[i*n + j] = B[(i+_koff)*n + j];
        //hipblasSetMatrix(_k, n, sizeof(float), hB, _k, dB, _k);
        hipblasSetVector(_k * n, sizeof(float), hB, 1, dB, 1);
        free(hB);
*/
          lda = transA ? _m : _k;
          ldb = transB ? _k : n;
          hipblasSetVector(_k * n, sizeof(float), B+(n*_koff), 1, dB, 1);
          hipblasSgemm(opA, opB, n, _m, _k, 1.0f, dB, ldb, dA, lda, factor, dC, n);
          cublasFree(dA);
          cublasFree(dB);

          _koff += _k;
          _k = k - _koff > _kmax ? _kmax : k - _koff;
          factor = 1.0f;
        }
      }
    }

    hipblasGetVector(_m*n, sizeof(float), dC, 1, C+(_moff*n), 1);
    cublasFree(dC);

    _moff += _m;
    _m = m - _moff > _mmax ? _mmax : m - _moff;
  }

  cublasShutdown();
}
/*
int main(int argc, char** argv) {
  struct timeval start, end; 
  double elapsed;

  float *A, *B, *C;
  int i, m, n, k;
 
  m = 20000;
  n = 20000;
  k = 20000;
  A = (float*)malloc(m*k*sizeof(float));
  B = (float*)malloc(k*n*sizeof(float));
  C = (float*)calloc(m*n,sizeof(float));

  for(i = 0; i < m*k; ++i)
    A[i] = 1;// + (i%2);
  for(i = 0; i < k*n; ++i)
    B[i] = 1;// + (i%2);

for(i=0;i<11;++i) {
  gettimeofday(&start,NULL);
  cusumma(0,0,m,n,k,A,B,C);
  gettimeofday(&end,NULL);

  elapsed = ((end.tv_sec*1000000 + end.tv_usec) - (start.tv_sec*1000000 + start.tv_usec))/1000000.0;
  printf("%f %f %f %f\n", C[0], C[m-1], C[m*(n-1)], C[m*n-1]);
  printf("%f\n", elapsed);
}

  free(A);
  free(B);
  free(C);
}
*/
