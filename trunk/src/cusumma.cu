#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>
#include <stdio.h>

extern "C" void
cusumma(unsigned int transA, 
        unsigned int transB, 
        unsigned int m,
        unsigned int n,
        unsigned int k,
        float *A, 
        float *B, 
        float *C)
{
  float *hA, *hB, *hC, *dA, *dB, *dC;
  float m_opt, k_opt;
  int i, j, diff, offset, _kmax, _m, _k, tm, tk;
  char opA, opB;
  unsigned int gpu_mem;

  cublasInit();

  // get total free memory available to SUMMA
  hipMemGetInfo(&gpu_mem, NULL);

  // take 2MB off the top for CUBLAS working memory
  // this is a guess that seems to work, replace with actual numbers when known
  gpu_mem -= 2*1048576;

  // convert gpu_mem from bytes into matrix elements (floats) for simplicity
  gpu_mem /= sizeof(float);

/*
  // determine optimal partition configuration
  // assume C is whole
  tw    = ceil((m*k + k*n)/(1.0*s - m*n));
  // assume C is partitioned
  m_opt = (sqrt(4.0*m*k*gpu_mem + (k+m)*(k+m)*n*n) - (k+m)*n)/(2.0*k);
  k_opt = (1.0*k/m) * m_opt;
  _m    = floor(m_opt);
  _k    = floor(k_opt);
  tk    = ceil(1.0*k/_k);
  tm    = ceil(1.0*m/_m);
  tp    = tk + tk*tm + tm;

  plan  = (tw > 0 && tw < tp) ? SINGLE_PARTITION : DOUBLE_PARTITION;
*/

  // assumes input matrices are in row-major order
  opA = transA ? 'n' : 't';
  opB = transB ? 'n' : 't';

  // op(A) * op(A)
  if(A == B) { 
    // allocate and initialize result matrix, substract from total free memory
    cublasAlloc(m * n, sizeof(float), (void**)&dC);
    hipblasSetVector(m * n, sizeof(float), C, 1, dC, 1); 
    gpu_mem -= m * n;

    diff = gpu_mem - m*k;
    // A can fit entirely on the device
    if(diff > 0) {
      cublasAlloc(m * k, sizeof(float), (void**)&dA);
      hipblasSetVector(m * k, sizeof(float), A, 1, dA, 1);
        
      hipblasSgemm(opA, opB, m, n, k, 1.0f, dA, k, dA, k, 0.0f, dC, m);
      cublasFree(dA);

    } else {
      // tk: assume cols for now, if transA, will have to flip
      _kmax  = gpu_mem / m;

      offset = 0;
      _k     = _kmax;
      while(offset < k) {
        hA = (float*)malloc(m * _k * sizeof(float));
        cublasAlloc(m * _k, sizeof(float), (void**)&dA);

        for(i = 0; i < m; ++i)
          for(j = 0; j < _k; ++j)
            hA[i * _k + j] = A[i * k + j + offset];
        hipblasSetVector(m * _k, sizeof(float), hA, 1, dA, 1);
        free(hA);

        hipblasSgemm(opA, opB, m, n, _k, 1.0f, dA, _k, dA, _k, 1.0f, dC, m);
        cublasFree(dA);
        
        offset += _k;
        _k      = k - offset > _kmax ? _kmax : k - offset;
      }
    }
  } 
  // op(A) * op(B)
  else {
    cublasAlloc(m * n, sizeof(float), (void**)&dC);
    hipblasSetVector(m * n, sizeof(float), C, 1, dC, 1); 
    gpu_mem -= m * n;

    diff = gpu_mem - (m*k + k*n);
    if(diff > 0) {

      cublasAlloc(m * k, sizeof(float), (void**)&dA);
      hipblasSetVector(m * k, sizeof(float), A, 1, dA, 1);

      cublasAlloc(k * n, sizeof(float), (void**)&dB);
      hipblasSetVector(k * n, sizeof(float), B, 1, dB, 1);
    
      hipblasSgemm(opA, opB, m, n, k, 1.0f, dA, k, dB, k, 0.0f, dC, m);

      cublasFree(dA);
      cublasFree(dB);

    } else {

      // tk: handle transpose, currently assumes A * B'
      _kmax   = gpu_mem / (m + n);
      _k      = _kmax;
      offset  = 0;

      while(offset < k) {
        hA = (float*) malloc(m * _k * sizeof(float));
        cublasAlloc(m * _k, sizeof(float), (void**)&dA);
        hB = (float*) malloc(_k * n * sizeof(float));
        cublasAlloc(_k * n, sizeof(float), (void**)&dB);
 

        for(i = 0; i < m; ++i)
          for(j = 0; j < _k; ++j)
            hA[i*_k + j] = A[i*k + j + offset];
        hipblasSetVector(m * _k, sizeof(float), hA, 1, dA, 1);
        free(hA);

        for(i = 0; i < n; ++i)
          for(j = 0; j < _k; ++j)
            hB[i*_k + j] = B[i*n + j + offset];
        hipblasSetVector(_k * n, sizeof(float), hB, 1, dB, 1);
        free(hB);

        hipblasSgemm(opA, opB, m, n, _k, 1.0f, dA, _k, dB, _k, 1.0f, dC, m);
        cublasFree(dA);
        cublasFree(dB);
        
        offset += _k;
        _k = k - offset > _kmax ? _kmax : k - offset;
      }
    }
  }

  hC = (float*) calloc(m*n, sizeof(float));
  hipblasGetVector(m*n, sizeof(float), dC, 1, hC, 1);
  cublasFree(dC);

  for(i = 0; i < m; ++i)
    for(j = 0; j < n; ++j)
      C[i*n+j] = hC[j*m+i];
  free(hC);  

  cublasShutdown();
}

int main(int argc, char** argv) {
  float *A, *B, *C;
  int i, m, n, k;
 
  m = 1000;
  n = 2;
  k = 400000;
  A = (float*)malloc(m*k*sizeof(float));
  B = (float*)malloc(k*n*sizeof(float));
  C = (float*)malloc(m*n*sizeof(float));

  for(i = 0; i < m*k; ++i)
    A[i] = 1;
  for(i = 0; i < k*n; ++i)
    B[i] = 1;

  // tk: trans(A)*A doesn't work if A isn't square
  //     need to swap params inside routine, not here
  //     maybe just go with full cblas-style inputs
  cusumma(0,1,m,n,k,A,B,C);
  printf("%f %f %f\n", C[0], C[1], C[2]);
  free(A);
  free(B);
  free(C);
}
