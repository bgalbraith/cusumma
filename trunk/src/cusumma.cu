#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" void
cudaSUMMA(unsigned int transA, 
          unsigned int transB, 
          unsigned int m,
          unsigned int n,
          unsigned int k,
          float *A, 
          float *B, 
          float *C)
{
  float *hA, *hB, *hC, *dA, *dB, *dC;
  int i, j, diff, offset, kmax;
  char opA, opB;
  unsigned int gpu_mem;
  size_t a_mem, b_mem, col_mem, row_mem;

  cublasInit();

  // get total free memory available to SUMMA
  hipMemGetInfo(&gpu_mem, NULL);

  // take 1MB off the top for CUBLAS working memory
  // this is a guess that seems to work, replace with actual numbers when known
  gpu_mem -= 1048576;

  // allocate and initialize result matrix, substract from total free memory
  cublasAlloc(m * n, sizeof(float), (void**)&dC);
  hipblasSetVector(m * n, sizeof(float), C, 1, dC, 1); 
  gpu_mem -= m * n * sizeof(float);

  // assumes input matrices are in row-major order
  opA = transA ? 'n' : 't';
  opB = transB ? 'n' : 't';

  // op(A) * op(A)
  if(A == B) { 
    a_mem = A->size * sizeof(float);
    diff = gpu_mem - a_mem;
    // A can fit entirely on the device
    if(diff > 0) {
      cublasAlloc(m * k, sizeof(float), (void**)&dA);
      hipblasSetVector(m * k, sizeof(float), A, 1, dA, 1);
        
      k = transA ? A->rows : A->cols;
      hipblasSgemm(opA, opB, C->rows, C->cols, k, 1.0f, dA, A->cols, dA, A->cols, 0.0f, dC, C->rows);
      cublasFree(dA);

    } else {
      // tk: assume cols for now, if transA, will have to flip
      col_mem = A->rows * sizeof(float);
      kmax    = gpu_mem / col_mem;
      k       = kmax;
      offset  = 0;

      while(offset < A->cols) {
        hA = (float*) malloc(A->rows * k * sizeof(float));
        cublasAlloc(A->rows * k, sizeof(float), (void**)&dA);

        for(i = 0; i < A->rows; ++i)
          for(j = 0; j < k; ++j)
            hA[i*k + j] = A->data[i*A->cols + j + offset];
        hipblasSetVector(A->rows * k, sizeof(float), hA, 1, dA, 1);
        free(hA);

        hipblasSgemm(opA, opB, C->rows, C->cols, k, 1.0f, dA, k, dA, k, 1.0f, dC, C->rows);
        cublasFree(dA);
        
        offset += k;
        k = A->cols - offset > kmax ? kmax : A->cols - offset;
      }
    }

  } else {
    a_mem = A->size * sizeof(float);
    b_mem = B->size * sizeof(float);
    diff = gpu_mem - a_mem - b_mem;
    if(diff > 0) {

      cublasAlloc(A->size, sizeof(float), (void**)&dA);
      hipblasSetVector(A->size, sizeof(float), A->data, 1, dA, 1);

      cublasAlloc(B->size, sizeof(float), (void**)&dB);
      hipblasSetVector(B->size, sizeof(float), B->data, 1, dB, 1);
      
      k = transA ? A->rows : A->cols;
      hipblasSgemm(opA, opB, C->rows, C->cols, k, 1.0f, dA, A->cols, dB, B->cols, 1.0f, dC, C->rows);

      cublasFree(dA);
      cublasFree(dB);

    } else {
      // tk: handle transpose, currently assumes A * B'
      col_mem = A->rows * sizeof(float);
      row_mem = B->rows * sizeof(float);
      kmax    = gpu_mem / (col_mem + row_mem);
      k       = kmax;
      offset  = 0;

      while(offset < A->cols) {
        hA = (float*) malloc(A->rows * k * sizeof(float));
        cublasAlloc(A->rows * k, sizeof(float), (void**)&dA);
        hB = (float*) malloc(k * B->rows * sizeof(float));
        cublasAlloc(k * B->rows, sizeof(float), (void**)&dB);
 

        for(i = 0; i < A->rows; ++i)
          for(j = 0; j < k; ++j)
            hA[i*k + j] = A->data[i*A->cols + j + offset];
        hipblasSetVector(A->rows * k, sizeof(float), hA, 1, dA, 1);
        free(hA);

        for(i = 0; i < B->rows; ++i)
          for(j = 0; j < k; ++j)
            hB[i*k + j] = B->data[i*B->cols + j + offset];
        hipblasSetVector(k * B->rows, sizeof(float), hB, 1, dB, 1);
        free(hB);

        hipblasSgemm(opA, opB, C->rows, C->cols, k, 1.0f, dA, k, dB, k, 1.0f, dC, C->rows);
        cublasFree(dA);
        cublasFree(dB);
        
        offset += k;
        k = A->cols - offset > kmax ? kmax : A->cols - offset;
      }
    }
  }

  hC = (float*) calloc(C->size, sizeof(float));
  hipblasGetVector(C->size, sizeof(float), dC, 1, hC, 1);
  cublasFree(dC);

  for(i = 0; i < C->rows; ++i)
    for(j = 0; j < C->cols; ++j)
      C->data[i*C->cols+j] = hC[j*C->rows+i];
  free(hC);  

  cublasShutdown();
}
